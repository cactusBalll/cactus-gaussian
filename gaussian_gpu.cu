#include "hip/hip_runtime.h"
/// compute capabilities 以RTX4050为例，它支持8.9，则每个SM最多24个block
/// 24(block) * 64(thread) = 1536(SM)
/// 1536个thread
/// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#features-and-technical-specifications-technical-specifications-per-compute-capability
#include "common.h"
#include "gaussian.h"
#include <cassert>
#include <hip/hip_cooperative_groups.h>

/// 提供了更丰富的同步原语
/// 见https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#cooperative-groups
namespace cg = cooperative_groups;

/// @brief 卷积核 存储在常量存储区
__constant__ float kernel[KERNEL_SIZE];

bool check_err() {
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    printf("cuda error.\n");
    printf("err code: %d\n", static_cast<int>(err));
    printf("description: %s\n", hipGetErrorString(err));
    return false;
  }
  return true;
}

void set_kernel(const float *k) { hipMemcpyToSymbol(HIP_SYMBOL(kernel), k, KERNEL_SIZE); }
/// @brief 行方向线程块大小x
constexpr int ROW_BLOCK_DIM_X = 16;
/// @brief 行方向线程块大小y
constexpr int ROW_BLOCK_DIM_Y = 4;
/// @brief 每个线程在每行生成卷积结果数量
constexpr int ROW_RESULT_STEP = 8;
/// @brief 每个线程两侧分别需要加载的光环元素
constexpr int ROW_HALO_STEP = 1;
/// @brief 处理的“砖块”大小x
constexpr int ROW_TILE_DIM_X = ROW_RESULT_STEP * ROW_BLOCK_DIM_X;
/// @brief 处理的“砖块”大小y
constexpr int ROW_TILE_DIM_Y = ROW_BLOCK_DIM_Y;
/// @brief shared memory行大小
constexpr int ROW_LOAD_SIZE_X =
    (ROW_RESULT_STEP + 2 * ROW_HALO_STEP) * ROW_BLOCK_DIM_X;

__global__ void convolution_row_kernel(float *dest, float *src, int image_h,
                                       int image_w) {
  cg::thread_block cta = cg::this_thread_block();
  __shared__ float shared_data[ROW_BLOCK_DIM_Y][ROW_LOAD_SIZE_X];

  // 计算TILE起始位置偏移，x方向减去一个HALO_STEP距离，光圈元素
  // base_x已经加上了threadIdx.x的偏移
  const int base_x =
      (blockIdx.x * ROW_RESULT_STEP - ROW_HALO_STEP) * ROW_BLOCK_DIM_X +
      threadIdx.x;
  const int base_y = blockIdx.y * ROW_BLOCK_DIM_Y + threadIdx.y;

  // 直接修改形参，应用偏移
  dest += base_y * image_w + base_x;
  src += base_y * image_w + base_x;

// 加载中间数据
// 编译指令：循环展开
#pragma unroll
  for (int i = ROW_HALO_STEP; i < ROW_HALO_STEP + ROW_RESULT_STEP; i++) {
    // 如果不假定图片长宽被TILE整除，则中间也需要判断幽灵元素
    if (base_x + i * ROW_BLOCK_DIM_X >= 0 &&
        base_x + i * ROW_BLOCK_DIM_X < image_w) {
      shared_data[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X] =
          src[i * ROW_BLOCK_DIM_X];
    }
  }
  // 加载光环元素
  shared_data[threadIdx.y][threadIdx.x] = (base_x >= 0) ? src[0] : 0;
  constexpr int RIGHT_HALO_OFFSET =
      (ROW_HALO_STEP + ROW_RESULT_STEP) * ROW_BLOCK_DIM_X;
  shared_data[threadIdx.y][threadIdx.x + RIGHT_HALO_OFFSET] =
      (base_x + RIGHT_HALO_OFFSET < image_w) ? src[RIGHT_HALO_OFFSET] : 0;

  // 同步，相当于__syncthreads
  cg::sync(cta);
#pragma unroll
  for (int i = ROW_HALO_STEP; i < ROW_HALO_STEP + ROW_RESULT_STEP; i++) {
    // 执行卷积，每个thread计算了多个位置
    float sum = 0;
#pragma unroll
    for (int j = -RADIUS; j <= RADIUS; j++) {
      sum += kernel[RADIUS + j] *
             shared_data[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X + j];
    }
    // 在范围内则写入
    if (base_x + i * ROW_BLOCK_DIM_X >= 0 &&
        base_x + i * ROW_BLOCK_DIM_X < image_w) {
      dest[i * ROW_BLOCK_DIM_X] = sum;
    }
  }
}

constexpr int COLUMN_BLOCK_DIM_X = 16;
constexpr int COLUMN_BLOCK_DIM_Y = 8;
constexpr int COLUMN_RESULT_STEP = 8;
constexpr int COLUMN_HALO_STEP = 1;
constexpr int COLUMN_TILE_DIM_X = COLUMN_BLOCK_DIM_X;
constexpr int COLUMN_TILE_DIM_Y = COLUMN_BLOCK_DIM_Y * COLUMN_RESULT_STEP;
constexpr int COLUMN_LOAD_SIZE_Y =
    (COLUMN_RESULT_STEP + COLUMN_HALO_STEP * 2) * COLUMN_BLOCK_DIM_Y;
__global__ void convolution_column_kernel(float *dest, float *src, int image_h,
                                          int image_w) {
  cg::thread_block cta = cg::this_thread_block();
  // 这里是转置的
  __shared__ float shared_data[COLUMN_TILE_DIM_X][COLUMN_LOAD_SIZE_Y];
  // threadIdx.x递增+1 接合
  const int base_x = blockIdx.x * COLUMN_BLOCK_DIM_X + threadIdx.x;
  const int base_y = (blockIdx.y * COLUMN_RESULT_STEP - COLUMN_HALO_STEP) *
                         COLUMN_BLOCK_DIM_Y +
                     threadIdx.y;
  dest += base_y * image_w + base_x;
  src += base_y * image_w + base_x;

// 加载中间元素
#pragma unroll
  for (int i = COLUMN_HALO_STEP; i < COLUMN_HALO_STEP + COLUMN_RESULT_STEP;
       i++) {
    shared_data[threadIdx.x][threadIdx.y + i * COLUMN_BLOCK_DIM_Y] =
        src[i * COLUMN_BLOCK_DIM_Y * image_w];
  }
  // 加载光环元素,上面和下面
  shared_data[threadIdx.x][threadIdx.y] = (base_y > 0) ? src[0] : 0;
  constexpr int LOWER_HALO_OFFSET =
      (COLUMN_HALO_STEP + COLUMN_RESULT_STEP) * COLUMN_BLOCK_DIM_Y;
  shared_data[threadIdx.x][threadIdx.y + LOWER_HALO_OFFSET] =
      (base_y + LOWER_HALO_OFFSET < image_h) ? src[LOWER_HALO_OFFSET * image_w]
                                             : 0;
  cg::sync(cta);
// 在列上执行卷积
#pragma unroll
  for (int i = COLUMN_HALO_STEP; i < COLUMN_HALO_STEP + COLUMN_RESULT_STEP;
       i++) {
    float sum = 0;
#pragma unroll
    for (int j = -RADIUS; j <= RADIUS; j++) {
      sum += kernel[j + RADIUS] *
             shared_data[threadIdx.x][threadIdx.y + i * COLUMN_BLOCK_DIM_Y + j];
    }
    dest[i * COLUMN_BLOCK_DIM_Y * image_w] = sum;
  }
}

extern "C" void convolution_row_gpu(float *dest, float *src, int image_h,
                                    int image_w) {
  dim3 blocks_row(image_w / ROW_TILE_DIM_X, image_h / ROW_TILE_DIM_Y);
  dim3 threads_row(ROW_BLOCK_DIM_X, ROW_BLOCK_DIM_Y);
  convolution_row_kernel<<<blocks_row, threads_row>>>(dest, src, image_h,
                                                      image_w);
}
extern "C" void convolution_column_gpu(float *dest, float *src, int image_h,
                                       int image_w) {
  dim3 blocks_column(image_w / COLUMN_TILE_DIM_X, image_h / COLUMN_TILE_DIM_Y);
  dim3 threads_column(COLUMN_BLOCK_DIM_X, COLUMN_BLOCK_DIM_Y);
  convolution_column_kernel<<<blocks_column, threads_column>>>(
      dest, src, image_h, image_w);
}
extern "C" void gaussian_gpu(float *dest, float *src, int image_h, int image_w,
                             float sigma) {
  // 至少1个tile大
  assert(image_h > COLUMN_TILE_DIM_Y);
  assert(image_w > ROW_TILE_DIM_X);
  // 且必须是TILE的整数倍
  // 对于照片这通常是满足的如3072*4096，也可以填充0
  assert(image_h % COLUMN_TILE_DIM_Y == 0);
  assert(image_w % ROW_TILE_DIM_Y == 0);

  const int buffer_size = image_h * image_w * sizeof(float);

  // 设备上内存指针
  float *input_device = nullptr;
  float *med_device = nullptr;
  float *output_device = nullptr;

  hipMalloc(&input_device, buffer_size);
  hipMalloc(&med_device, buffer_size);
  hipMalloc(&output_device, buffer_size);
  // 检查错误
  if (!check_err()) {
    hipFree(input_device);
    hipFree(med_device);
    hipFree(output_device);
    return;
  }
  // 从内存复制输入到设备
  hipMemcpy(input_device, src, buffer_size, hipMemcpyHostToDevice);
  // 检查错误
  if (!check_err()) {
    hipFree(input_device);
    hipFree(med_device);
    hipFree(output_device);
    return;
  }

  auto kernel_vec = make_gaussian_kernel_1d(sigma);
  // 高斯卷积核是对称的，分离的两个1维卷积使用同样的卷积核
  set_kernel(kernel_vec.data());
  convolution_row_gpu(med_device, input_device, image_h, image_w);
  convolution_column_gpu(output_device, med_device, image_h, image_w);

  // 从设备把结果复制回内存
  hipMemcpy(dest, output_device, buffer_size, hipMemcpyDeviceToHost);
  check_err();
  hipFree(input_device);
  hipFree(med_device);
  hipFree(output_device);
}